// Starter CUDA Intro
// Based on Fireship's CUDA in 100 seconds
// 9/30/2024 (Happy Birthday :D)


#include <hip/hip_runtime.h>
#include <stdio.h>

// Use global specifier to define function or CUDA kernel that runs on GPU
__global__ void add(int* a, int* b, int* c)
{
    // Adds 2 vectors (a and b) together

    // Because we're doing calculations in parallel, we need to calculate the
    // global index of the thread and the block that we're working on 
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] + b[i];
}

// Use managed specifier to tell CUDA that this data can be accessed from the 
// CPU and GPU without having to manually copy data between them
__managed__ int vector_a[400], vector_b[400], vector_c[400];

// main function for the CPU that runs the CUDA kernel
int main()
{
    // Populates our vectors with data
    for (int i = 0; i < 400; i++)
    {
        vector_a[i] = i;

        vector_b[i] = 400+i;
    }

    // Triple brackets allow us to configure the CUDA kernel launch to control how
    // many blocks and how many threads per block to use
    // Syntax: <<<blocks, threads per block>>>
    add<<<4, 100>>>(vector_a, vector_b, vector_c);

    // Pauses execution and waits for it to complete on the GPU
    hipDeviceSynchronize();

    // Once completed on the GPU, we add up all of the vectors in vector c
    int result_sum = 0;
    for (int i = 0; i < 400; i++)
    {
        result_sum += vector_c[i];
        printf("Value at index %d: %d\n", i, vector_c[i]);
    }

    printf("Results: sum = % d", result_sum);
}